#include "hip/hip_runtime.h"
%%writefile cuda.cu
/*
Reference: Pacheco, P. S., Malensek, M. An Introduction To Parallel Programming
Chapter 6
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

//kernel function
__global__ void Vec_add(
 const float x[],
 const float y[],
 float z[],
 const int n
)
{
    int my_index = blockDim.x * blockIdx.x + threadIdx.x;
    if (my_index < n) {
        z[my_index] = x[my_index] + y[my_index];
    }
}

void Get_args(
    const int argc,
    char* argv[],
    int* n_p,//out,
    int* blk_ct_p,//out
    int* th_per_blk_p, //out
    char* i_g
) {
    if (argc != 5) {
        printf("There should be at most 5 arguments");
    }
    *n_p = strtol(argv[1], NULL, 10); //
    *blk_ct_p = strtol(argv[2], NULL, 10); //get number of blocks
    *th_per_blk_p = strtol(argv[3], NULL, 10); //get number of threads per block
    *i_g = argv[4][0];
    int total_threads = (*blk_ct_p)*(*th_per_blk_p);
    if (*n_p > total_threads) {
        printf("Number of elements %d > number of total threads %d", *n_p, total_threads);
        exit(1);
    }
    return;
}

void Init_vectors(
    float* x,
    float* y,
    const int n,
    const char i_g
) {
    for (int i = 0; i < n; ++i){
        x[i] = i*10.0;
        y[i] = i*5.0;
    }
}

double Two_norm_diff(
    const float z[],
    const float cz[],
    const int n) {
        double diff, sum =0.0;
        for (int i = 0; i < n; ++i) {
            diff = z[i] - cz[i];
            sum += diff*diff;
        }
        return sqrt(sum);
}

void Free_cuda_vectors(
    float* x,
    float* y,
    float* z
) {
    hipFree(x);
    hipFree(y);
    hipFree(z);
}

void Allocate_cuda_vectors(
    float** x,
    float** y,
    float** z,
    int n
) {
    hipMalloc(x, n*sizeof(float));
    hipMalloc(y, n*sizeof(float));
    hipMalloc(z, n*sizeof(float));
}

void Allocate_host_vectors(
    float** x,
    float** y,
    float** z,
    float** cz,
    int n
) {
    *x = (float*) malloc(n*sizeof(float));
    *y = (float*) malloc(n*sizeof(float));
    *z = (float*) malloc(n*sizeof(float));
    *cz = (float*) malloc(n*sizeof(float));
}

void Free_host_vectors(
    float* x,
    float* y,
    float* z,
    float* cz
) {
    free(x);
    free(y);
    free(z);
    free(cz);
}

void Serial_vec_add(
    const float* x,
    const float* y,
    float* z,
    const int n
) {
    for (int i = 0; i < n; ++i) {
        z[i] = x[i] + y[i];
    }
}

int main(int argc, char* argv[]) {
    //Assuming none unified memory
    int n, threadPerBlock, blockCount;
    char i_g; /*are x and y user input or random*/
    float *hx, *hy, *hz, *cz; //host array
    float *dx, *dy, *dz; // device arrays
    double diff_norm;

    Get_args(argc, argv, &n, &blockCount, &threadPerBlock, &i_g);
    Allocate_cuda_vectors(&dx,&dy,&dz, n);
    Allocate_host_vectors(&hx,&hy,&hz,&cz,n);
    Init_vectors(hx,hy,n,i_g);
    
    hipMemcpy(dx, hx, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, n*sizeof(float), hipMemcpyHostToDevice);

    Vec_add <<<blockCount, threadPerBlock>>>(dx,dy,dz,n);
    
    //wait and copy result from GPU device to host
    hipMemcpy(hz,dz, n*sizeof(float), hipMemcpyDeviceToHost);

    Serial_vec_add(hx,hy,cz,n);
    diff_norm = Two_norm_diff(hz, cz, n);
    printf("two norm of difference between host and ");
    printf("device = %e\n", diff_norm);

    //cleanup
    Free_cuda_vectors(dx,dy,dz);
    Free_host_vectors(hx, hy, hz, cz);

    return 0;
}